#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_error_macros.h"

//#define USE_FLOAT
#ifdef USE_FLOAT
using ArethmT = float;
#else 
using ArethmT = double;
#endif


#include "matrix.h"

#include <stdio.h>
#include <stdlib.h>
#include <initializer_list>

float cublassTime = 0.f;
float basicTime = 0.f;
float optimisedTime = 0.f;




Matrix GetRandomInputMatrix();

Matrix GetCustomInputMatrix() {
	return std::move(Matrix(4, {
		0,  1,  2,  3,
		10, 11, 12, 13,
		20, 21, 22, 23
	}));
}

// Returns milliseconds for the time Function f took.
template<typename Function>
float CountTime(Function&& f) {
	hipEvent_t start, stop;
	hipEventCreate(&start); CE;
	hipEventCreate(&stop); CE;

	hipEventRecord(start); CE;

	f();

	hipEventRecord(stop); CE;
	hipEventSynchronize(stop); CE;

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop); CE;
	return milliseconds;
}

// Integer division + ceil if remainder
int div_ceil(int lhs, int rhs) {
	return lhs / rhs + (lhs % rhs == 0 ? 0 : 1);
}

Matrix cublas_Tmultiply(Matrix& inputMatrix) {
	const ArethmT alpha = 1;
	const ArethmT beta = 0;

	Matrix result(inputMatrix.cols, inputMatrix.cols);
	result.AllocDevice();

	inputMatrix.IntoDevMatrix_ColMajor();
	
	hipblasHandle_t handle;
	auto status = hipblasCreate(&handle); CBE(status);

	int N = inputMatrix.cols;
	int M = inputMatrix.rows;

	cublassTime = CountTime([&]() {
#ifdef USE_FLOAT
		status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, M, &alpha,
							 inputMatrix.dev_data, M, inputMatrix.dev_data, M, &beta, result.dev_data, N);
#else
		status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, M, &alpha,
							 inputMatrix.dev_data, M, inputMatrix.dev_data, M, &beta, result.dev_data, N);
#endif
	});
	CBE(status);
	hipblasDestroy(handle);


	result.FromDevMatrix_ColMajor();

	result.FreeDevice();
	inputMatrix.FreeDevice();

	return std::move(result);
}

#define AT(row, col, nr_cols) ((row) * (nr_cols) + (col))


//
// SIMPLE CUDA
//
__global__ void basic_dev_Tmultiply(int nr_rows, int nr_cols, ArethmT* src, ArethmT* result)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= nr_cols || col >= nr_cols) {
		return;
	}
	
	ArethmT elem = 0;

	for (int i = 0; i < nr_rows; ++i) {
		elem += src[AT(i, row, nr_cols)] * src[AT(i, col, nr_cols)];
	}

	result[AT(row, col, nr_cols)] = elem;

}

Matrix basic_Tmutliply(Matrix& input) {
	Matrix result(input.cols, input.cols);
	result.AllocDevice();

	input.IntoDevMatrix();

	constexpr int Threads = 32;
	const int GridSize = div_ceil(input.cols, Threads);

	dim3 block(Threads, Threads);
	dim3 grid(GridSize, GridSize);

	basicTime = CountTime([&]() {
		basic_dev_Tmultiply << <grid, block>> > (input.rows, input.cols, input.dev_data, result.dev_data);
	});

	result.FromDevMatrix();

	input.FreeDevice();
	result.FreeDevice();

	return result;
}

Matrix opt_Tmutliply(Matrix& input);


int main() {
//	Matrix inputMatrix = GetCustomInputMatrix();
	Matrix inputMatrix = GetRandomInputMatrix();

	hipSetDevice(0); CE;

	Matrix cublasResult;
	cublasResult = cublas_Tmultiply(inputMatrix);
	printf("Result: \n");
	
	Matrix basicCudaResult;
	basicCudaResult = basic_Tmutliply(inputMatrix);

	if (!cublasResult.IsNearlyEqual(basicCudaResult)) {
		printf("basic cuda had different result than cublass");
	}

	
	Matrix optCudaResult;
	optCudaResult = opt_Tmutliply(inputMatrix);


	if (cublasResult.IsNearlyEqual(optCudaResult)) {
		printf("Opt cuda was correct.\n");
	}
	else {
		printf("Opt cuda was different: cublas:\n");
		cublasResult.Print();
		printf("==== src:\n");
		inputMatrix.Print();
		printf("==== cuda:\n");
		optCudaResult.Print();
	}

	printf("cuBLASS: %4.4f ms\n", cublassTime);
	printf("basic  : %4.4f ms\n", basicTime);
	printf("optimis: %4.4f ms\n", optimisedTime);


	hipDeviceReset();
	return 0;
}


//
// OPTIMISED CUDA
//

//
// Optimisation Notes: 
// 1. T(A) * A is symmetrical
// 2. 
//
//
// Testing:
// 1. Fill our 11GB GPU buffer without crashing the the OS & driver. 
//    Check if offsets can be applied to make 1 buffer with multiple cuda memcopies.
// 

enum class DebugOutput {
	Result = 0,
	RowCol = 1,
	Thread = 2,
	Block = 3
};


Matrix GetRandomInputMatrix() {
	constexpr int TestRows = 5120;
	constexpr int TestCols = 2560;

	Matrix inputMatrix(TestCols, TestRows);
	inputMatrix.AllocHost();

	for (int i = 0; i < inputMatrix.Size(); ++i) {
		inputMatrix.data[i] = i % TestCols;// std::rand() % 64;
	}

	return std::move(inputMatrix);
}



// These provide 100% occupancy on our GPU TU102
constexpr int BLOCK_WIDTH = 16;
constexpr int ELEM_PT = 2; // SqrtRoot of elements calculated per thread.
constexpr int TILE_WIDTH = BLOCK_WIDTH * ELEM_PT;




__device__ void dev_WriteResult(int nr_rows, int nr_cols, ArethmT* src, ArethmT* output, int res_row, int res_col)
{
	const int bx = blockIdx.x;
	const int by = blockIdx.y;

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	ArethmT result = 0.0;

	const int block_start_x = bx * TILE_WIDTH;
	const int block_start_y = by * TILE_WIDTH;


	// transpose(A) * A in practice means we multiply between 2 columns. With this as the base assumption the rest of
	// the code in this function may use the terms column, column_1 and column_2 meaning the columns from the "src"
	// that are needed to be multiplyied. This makes the code simpler and easier to read.

	__shared__ ArethmT sm_col_onY[TILE_WIDTH][TILE_WIDTH];
	__shared__ ArethmT sm_col_onX[TILE_WIDTH][TILE_WIDTH];

	ArethmT reg_onX[2][ELEM_PT][ELEM_PT];
	ArethmT reg_onY[2][ELEM_PT][ELEM_PT];
	
	ArethmT reg_accum[ELEM_PT][ELEM_PT];



	// prepare first loop data...

	int m;
	for (m = 0; m < nr_rows / TILE_WIDTH; ++m) {

		// Copying here is a bit different than the usual A*B multiplication
		// We copy blocks from the required columns directly as squares
		sm_col_onX[ty][tx] = src[AT(m * TILE_WIDTH + tx, block_start_x + tx, nr_cols)];
		sm_col_onY[ty][tx] = src[AT(m * TILE_WIDTH + tx, block_start_y + tx, nr_cols)];

		__syncthreads();


		for (int k = 0; k < TILE_WIDTH; ++k) {
			result += sm_col_onX[k][tx] * sm_col_onY[k][ty];
		}


		__syncthreads();
	}

	output[AT(res_row, res_col, nr_cols)] = reg_onX[0][tx][ty] + reg_accum[tx][ty];
	output[AT(res_col, res_row, nr_cols)] = result;
}


// Prepare loop
// FOR: ...
//		copy NEXT partition from global -> shared  // GLOBAL -> SHARED: Threads*2
//		__sync()
//		FOR TILE:
//			copy from SHARED -> REGISTERS (prefetch)
//			Multiply 2*2 from registers[0]
//			Swap Register Buffer
//		__sync()
//




__global__ void opt_dev_Tmultiply_NoOdd(int nr_rows, int nr_cols, ArethmT* src, ArethmT* result)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	// Triangular due to symmetry of t(A) * A && border checks
	if (blockIdx.y > blockIdx.x) {
		return;
	}

	dev_WriteResult(nr_rows, nr_cols, src, result, row, col);
	
	DebugOutput debugOutput = DebugOutput::Result;
	if (debugOutput == DebugOutput::Result) {
		
	}
	else if (debugOutput == DebugOutput::RowCol) {
		result[AT(row, col, nr_cols)] = row + col * 100;
	}
	else if (debugOutput == DebugOutput::Thread) {
		result[AT(row, col, nr_cols)] = threadIdx.x + threadIdx.y * 100;
	}
	else if (debugOutput == DebugOutput::Block) {
		result[AT(row, col, nr_cols)] =  blockIdx.x + blockIdx.y * 100;
	}

}

Matrix opt_Tmutliply(Matrix& input) {
	Matrix result(input.cols, input.cols);
	result.AllocDevice();

	input.IntoDevMatrix();

	constexpr int Threads = BLOCK_WIDTH;
	const int GridSize = div_ceil(input.cols, TILE_WIDTH);

	dim3 block(Threads, Threads);
	dim3 grid(GridSize, GridSize);
	

	optimisedTime = CountTime([&]() {
		opt_dev_Tmultiply_NoOdd <<<grid, block>>> (input.rows, input.cols, input.dev_data, result.dev_data);
	});
	result.FromDevMatrix();

	input.FreeDevice();
	result.FreeDevice();

	return result;
}
