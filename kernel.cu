#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_error_macros.h"
#include "matrix.h"

#include <stdio.h>
#include <stdlib.h>
#include <initializer_list>

Matrix cublas_Tmultiply(Matrix& inputMatrix) {
	const float alpha = 1;
	const float beta = 0;

	Matrix result(inputMatrix.cols, inputMatrix.cols);
	result.AllocDevice();

	inputMatrix.IntoDevMatrix_ColMajor();
	
	hipblasHandle_t handle;
	auto status = hipblasCreate(&handle); CBE(status);

	int N = inputMatrix.cols;
	int M = inputMatrix.rows;
	status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, M, &alpha,
							  inputMatrix.dev_data, M, inputMatrix.dev_data, M, &beta, result.dev_data, N); CBE(status);
	hipblasDestroy(handle);


	result.FromDevMatrix_ColMajor();

	result.FreeDevice();
	inputMatrix.FreeDevice();

	return std::move(result);
}

#define AT(row, col, nr_cols) (row * nr_cols + col)



__global__ void basic_dev_Tmultiply(int nr_rows, int nr_cols, float* src, float* result)
{
	int row = threadIdx.x / nr_cols;
	int col = threadIdx.x % nr_cols;
	
	float sum = 0;

	for (int i = 0; i < nr_cols; ++i) {
		sum += src[AT(i, row, nr_cols)] * src[AT(i, col, nr_cols)];
	}

	result[AT(row, col, nr_cols)] = sum + 1;

//	Index Debugger
//	result[AT(row, col, nr_cols)] = row + 10 * col;
}

Matrix basic_Tmutliply(Matrix& input) {
	Matrix result(input.cols, input.cols);
	result.AllocDevice();

	input.IntoDevMatrix();

	basic_dev_Tmultiply<<<1, result.Size()>>>(input.rows, input.cols, input.dev_data, result.dev_data);

	result.FromDevMatrix();

	input.FreeDevice();
	result.FreeDevice();

	return result;
}


int main() {
	Matrix inputMatrix(4, {
		0,  1,  2,  3,
		10, 11, 12, 13,
		20, 21, 22, 23
	});

	hipSetDevice(0); CE;

	Matrix cublasResult;
	cublasResult = cublas_Tmultiply(inputMatrix);
	
	Matrix basicCudaResult;
	basicCudaResult = basic_Tmutliply(inputMatrix);

	if (cublasResult.IsNearlyEqual(basicCudaResult)) {
		printf("Result: \n");
		cublasResult.Print();
	}
	else {
		printf("Different results: \ncublas:\n");
		cublasResult.Print();
		printf("simple cuda:\n");
		basicCudaResult.Print();
	}



	hipDeviceReset();
	return 0;
}
