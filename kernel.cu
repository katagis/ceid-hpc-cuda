#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_error_macros.h"
#include "matrix.h"

#include <stdio.h>
#include <stdlib.h>
#include <initializer_list>

float cublassTime = 0.f;
float basicTime = 0.f;
float optimisedTime = 0.f;


Matrix GetRandomInputMatrix();

Matrix GetCustomInputMatrix() {
	return std::move(Matrix(4, {
		0,  1,  2,  3,
		10, 11, 12, 13,
		20, 21, 22, 23
	}));
}

// Returns milliseconds for the time Function f took.
template<typename Function>
float CountTime(Function&& f) {
	hipEvent_t start, stop;
	hipEventCreate(&start); CE;
	hipEventCreate(&stop); CE;

	hipEventRecord(start); CE;

	f();

	hipEventRecord(stop); CE;
	hipEventSynchronize(stop); CE;

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop); CE;
	return milliseconds;
}

// Integer division + ceil if remainder
int div_ceil(int lhs, int rhs) {
	return lhs / rhs + (lhs % rhs == 0 ? 0 : 1);
}

Matrix cublas_Tmultiply(Matrix& inputMatrix) {
	const double alpha = 1;
	const double beta = 0;

	Matrix result(inputMatrix.cols, inputMatrix.cols);
	result.AllocDevice();

	inputMatrix.IntoDevMatrix_ColMajor();
	
	hipblasHandle_t handle;
	auto status = hipblasCreate(&handle); CBE(status);

	int N = inputMatrix.cols;
	int M = inputMatrix.rows;

	cublassTime = CountTime([&]() {
		status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, M, &alpha,
							 inputMatrix.dev_data, M, inputMatrix.dev_data, M, &beta, result.dev_data, N);
	});
	CBE(status);
	hipblasDestroy(handle);


	result.FromDevMatrix_ColMajor();

	result.FreeDevice();
	inputMatrix.FreeDevice();

	return std::move(result);
}

#define AT(row, col, nr_cols) (row * nr_cols + col)
#define AT_T(row, col, nr_rows) (row + col * nr_rows)


//
// SIMPLE CUDA
//
__global__ void basic_dev_Tmultiply(int nr_rows, int nr_cols, double* src, double* result)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= nr_cols || col >= nr_cols) {
		return;
	}
	
	double elem = 0;

	for (int i = 0; i < nr_rows; ++i) {
		elem += src[AT(i, row, nr_cols)] * src[AT(i, col, nr_cols)];
	}

	result[AT(row, col, nr_cols)] = elem;

}

Matrix basic_Tmutliply(Matrix& input) {
	Matrix result(input.cols, input.cols);
	result.AllocDevice();

	input.IntoDevMatrix();

	constexpr int Threads = 32;
	const int GridSize = div_ceil(input.cols, Threads);

	dim3 block(Threads, Threads);
	dim3 grid(GridSize, GridSize);

	basicTime = CountTime([&]() {
		basic_dev_Tmultiply << <grid, block>> > (input.rows, input.cols, input.dev_data, result.dev_data);
	});

	result.FromDevMatrix();

	input.FreeDevice();
	result.FreeDevice();

	return result;
}

Matrix opt_Tmutliply(Matrix& input);


int main() {
//	Matrix inputMatrix = GetCustomInputMatrix();
	Matrix inputMatrix = GetRandomInputMatrix();

	hipSetDevice(0); CE;

	Matrix cublasResult;
	cublasResult = cublas_Tmultiply(inputMatrix);
	printf("Result: \n");

	Matrix basicCudaResult;
	basicCudaResult = basic_Tmutliply(inputMatrix);

	if (!cublasResult.IsNearlyEqual(basicCudaResult)) {
		printf("basic cuda had different result than cublass");
	}

	
	Matrix optCudaResult;
	optCudaResult = opt_Tmutliply(inputMatrix);

	if (cublasResult.IsNearlyEqual(optCudaResult)) {
		printf("Opt cuda was correct.\n");
	}
	else {
		printf("Opt cuda was different:\n");
		printf("cublas:\n");
		cublasResult.Print();

		printf("src:\n");
		inputMatrix.Print();

		printf("cuda:\n");
		optCudaResult.Print();
	}


	printf("cuBLASS: %4.4f ms\n", cublassTime);
	printf("basic  : %4.4f ms\n", basicTime);
	printf("optimis: %4.4f ms\n", optimisedTime);


	hipDeviceReset();
	return 0;
}

Matrix GetRandomInputMatrix() {
	constexpr int TestRows = 5120;
	constexpr int TestCols = 2560;

	Matrix inputMatrix(TestCols, TestRows);
	inputMatrix.AllocHost();

	for (int i = 0; i < inputMatrix.Size(); ++i) {
		inputMatrix.data[i] = i % TestCols;
	}

	for (int i = 0; i < TestRows; ++i) {
		inputMatrix.data[i] = i * 10;
	}


	return std::move(inputMatrix);
}


//
// OPTIMISED CUDA
//

//
// Optimisation Notes: 
// 1. T(A) * A is symmetrical
// 2. 
//

// Optimisations done based on:
// https://www.seas.upenn.edu/~cis565/Lectures2011S/Lecture12.pdf // Prefetch 
// https://ecatue.gitlab.io/gpu2018/pages/Cookbook/matrix_multiplication_cuda.html#5 // Shared memory + Bank conflicts

enum class DebugOutput {
	Result = 0,
	RowCol = 1,
	Thread = 2,
	Block = 3
};

constexpr DebugOutput debugOutput = DebugOutput::Result;

constexpr int BLOCK_SIZE = 32;
constexpr int TILE_SIZE = 32;

__device__ void dev_WriteResult(int nr_rows, int nr_cols, double* src, double* output, int res_row, int res_col)
{
	const int bx = blockIdx.x;
	const int by = blockIdx.y;

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int block_start_x = bx * TILE_SIZE;
	const int block_start_y = by * TILE_SIZE;

	// transpose(A) * A in practice means we multiply between 2 columns. With this as the base assumption the rest of
	// the code in this function may use the terms column, column_1 and column_2 meaning the columns from the "src"
	// that are needed to be multiplyied. This makes the code simpler and easier to read.
	
	// Also note that A is in Column-Major

	__shared__ double sm_col_onY[TILE_SIZE][TILE_SIZE];
	__shared__ double sm_col_onX[TILE_SIZE][TILE_SIZE];

	double result = 0.0;


	// prepare first loop data...

	// Column to copy for this thread based on the C tile's x
	const int col_x = block_start_x + tx;
	
	// Column to copy for this thread based on the C tile's y
	const int col_y = block_start_y + tx;

	int m;
	for (m = 0; m < nr_rows; m += TILE_SIZE) {

		// Copying here is a bit different than the usual A*B multiplication
		// We copy blocks from the required columns directly as squares
		sm_col_onX[ty][tx] = src[AT_T(m + ty, col_x, nr_rows)];
		sm_col_onY[ty][tx] = src[AT_T(m + ty, col_y, nr_rows)];

		__syncthreads();

#pragma unroll
		for (int k = 0; k < TILE_SIZE; ++k) {
			result += sm_col_onX[k][tx] * sm_col_onY[k][ty];
		}


		__syncthreads();
	}

	output[AT(res_row, res_col, nr_cols)] = result;
	output[AT(res_col, res_row, nr_cols)] = result;
}



__global__ void opt_dev_Tmultiply(int nr_rows, int nr_cols, double* src, double* result)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (blockIdx.x > blockIdx.y) {
		return;
	}

	dev_WriteResult(nr_rows, nr_cols, src, result, row, col);


	if (debugOutput == DebugOutput::Result) {

	}
	else if (debugOutput == DebugOutput::RowCol) {
		result[AT(row, col, nr_cols)] = row + col * 100;
	}
	else if (debugOutput == DebugOutput::Thread) {
		result[AT(row, col, nr_cols)] = threadIdx.x + threadIdx.y * 100;
	}
	else if (debugOutput == DebugOutput::Block) {
		result[AT(row, col, nr_cols)] =  blockIdx.x + blockIdx.y * 100;
	}

}

Matrix opt_Tmutliply(Matrix& input) {
	Matrix result(input.cols, input.cols);
	result.AllocDevice();

	input.IntoDevMatrix_ColMajor();


	constexpr int Threads = BLOCK_SIZE;
	const int GridSize = div_ceil(input.cols, TILE_SIZE);

	dim3 block(Threads, Threads);
	dim3 grid(GridSize, GridSize);
	

	optimisedTime = CountTime([&]() {
		opt_dev_Tmultiply <<<grid, block>>> (input.rows, input.cols, input.dev_data, result.dev_data);
	});
	result.FromDevMatrix();

	input.FreeDevice();
	result.FreeDevice();

	return result;
}
